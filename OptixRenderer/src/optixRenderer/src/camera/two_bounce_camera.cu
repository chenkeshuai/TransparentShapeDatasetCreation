#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "structs/prd.h"
#include "random.h"

using namespace optix;


rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        cameraU, , );
rtDeclareVariable(float3,        cameraV, , );
rtDeclareVariable(float3,        cameraW, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(int,           max_depth, , );

rtBuffer<float3, 2>              normal1_buffer;
rtBuffer<float3, 2>              depth1_buffer;
rtBuffer<float, 2>              mask1_buffer;
rtBuffer<float3, 2>              normal2_buffer;
rtBuffer<float3, 2>              depth2_buffer;
rtBuffer<float, 2>              mask2_buffer;

rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );
rtDeclareVariable(unsigned int, sqrt_num_samples, , );
rtDeclareVariable(unsigned int, rr_begin_depth, , );
rtDeclareVariable(int, cameraMode, , );
rtDeclareVariable(unsigned int, initSeed, , );

RT_PROGRAM void pinhole_camera()
{
    size_t2 screen = normal1_buffer.size();
    
    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;

    unsigned int seed = tea<32>( 
        ( (initSeed)*(screen.x*launch_index.y+launch_index.x) + initSeed ), 
        ( (screen.y * launch_index.x + launch_index.y) * initSeed ) );
    
    float hitCount = 0.0;

    float mask1 = 0.0;
    float3 normal1 = make_float3(0.0f);
    float3 depth1 = make_float3(0.0f);

    float mask2 = 0.0;
    float3 normal2 = make_float3(0.0f);
    float3 depth2 = make_float3(0.0f);

    do{
        // Sample pixel using jittering
        float3 ray_origin = eye;
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed) );
        float2 d = pixel + jitter*jitter_scale;

        
        float3 ray_direction;
        if(cameraMode == 0){
            ray_direction = normalize(d.x*cameraU + d.y*cameraV + cameraW);
        }
        else{
            float3 axisZ = normalize(cameraW );
            float3 axisX = normalize(cameraU );
            float3 axisY = normalize(cameraV );

            d.x = (d.x > 1.0f) ? 1.0f : d.x;
            d.y = (d.y > 1.0f) ? 1.0f : d.y;
            d.x = (d.x < -1.0f) ? -1.0f : d.x;
            d.y = (d.y < -1.0f) ? -1.0f : d.y;

            float phi = d.x * M_PIf;
            float theta;
            if(cameraMode == 1){
                theta = 0.5f * (-d.y + 1.0f) * M_PIf;
            }
            else if(cameraMode == 2){
                theta = 0.25f * (-d.y + 1.0f) * M_PIf;
            }
            ray_direction = normalize(
                    sinf(theta) * cosf(phi) * axisX 
                    + sinf(theta) * sinf(phi) * axisY 
                    + cosf(theta) * axisZ
                    );
        }

        // Initialze per-ray data
        TwoBounce_data prd;
        prd.done = false;
        prd.seed = seed;
        prd.depth = 0;
        prd.direction = ray_direction;

        prd.isHit = false;
        prd.mask1 = 0.0;
        prd.normal1 = make_float3(0.0f);
        prd.depth1 = make_float3(0.0f);
        prd.mask2 = 0.0;
        prd.normal2 = make_float3(0.0f);
        prd.depth2 = make_float3(0.0f);

        // Each iteration is a segment of the ray path.  The closest hit will
        // return new segments to be traced here.
        for(;;)
        {
            Ray ray(ray_origin, ray_direction, 0, scene_epsilon);
            rtTrace(top_object, ray, prd);
 
            prd.depth++; 
           
            // Hit the light source or exceed the max depth
            if(prd.done || prd.depth >= max_depth) 
                break; 
            
            // Update ray data for the next path segment
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }
        mask1 += prd.mask1;
        mask2 += prd.mask2;

        if(prd.isHit == true){
            hitCount += 1;
            normal1 += prd.normal1;
            depth1 += prd.depth1; 
            normal2 += prd.normal2; 
            depth2 += prd.depth2;
        }

        seed = prd.seed;
    } while (--samples_per_pixel);

    // Update the output buffer
    unsigned sampleNum = sqrt_num_samples * sqrt_num_samples;
    mask1_buffer[launch_index ] = mask1 / sampleNum;
    if(hitCount != 0){
        if(normal1.x != 0.0 || normal1.y != 0.0 || normal1.z != 0.0 ){
            normal1_buffer[launch_index ] = normalize(normal1 );
        }
        else{
            normal1_buffer[launch_index ] = make_float3(0.0);
        }
        depth1_buffer[launch_index ] = depth1 / hitCount;
    }
    else{
        normal1_buffer[launch_index ] = make_float3(0.0 );
        depth1_buffer[launch_index] = make_float3(0.0 );
    }

    mask2_buffer[launch_index ] = mask2 / sampleNum;
    if(hitCount != 0){
        if(normal2.x != 0.0 || normal2.y != 0.0 || normal2.z != 0.0){
            normal2_buffer[launch_index ] = normalize(normal2 );
        }
        else{
            normal2_buffer[launch_index ] = make_float3(0.0 );
        }
        depth2_buffer[launch_index ] = depth2 / hitCount;
    }
    else{
        normal2_buffer[launch_index ] = make_float3(0.0 );
        depth2_buffer[launch_index ] = make_float3(0.0);
    }
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  normal1_buffer[launch_index] = bad_color;
  depth1_buffer[launch_index] = bad_color;
  normal2_buffer[launch_index] = bad_color;
  depth2_buffer[launch_index] = bad_color;
}
