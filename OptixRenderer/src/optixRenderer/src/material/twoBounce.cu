#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "structs/prd.h"
#include "random.h"
#include "commonStructs.h"
#include "lightStructs.h"
#include <vector> 

using namespace optix; 

rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, tangent_direction, attribute tangent_direction, );
rtDeclareVariable(float3, bitangent_direction, attribute bitangent_direction, );

rtDeclareVariable( float3, texcoord, attribute texcoord, );
rtDeclareVariable( float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(optix::Ray, ray,   rtCurrentRay, );
rtDeclareVariable(TwoBounce_data, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );
rtDeclareVariable(float, scene_epsilon, , );

// Normal 
rtTextureSampler<float4, 2> normalMap;
rtDeclareVariable( int, isNormalTexture, , );

// Material 
rtDeclareVariable(float, intIOR, , );
rtDeclareVariable(float, extIOR, , );

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable( float3, cameraU, , );
rtDeclareVariable( float3, cameraV, , );
rtDeclareVariable( float3, cameraW, , );

// Geometry Group
rtDeclareVariable( rtObject, top_object, , );

RT_CALLABLE_PROGRAM void sample(unsigned& seed, 
        float3 N, const float3& V, 
        float3& attenuation, float3& direction, float& pdfSolid)
{
}


RT_PROGRAM void closest_hit_radiance()
{
    const float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    const float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );

    float3 V = normalize(-ray.direction );
    
    float3 N;
    if( isNormalTexture == 0){
        N = world_shading_normal;
    }
    else{
        N = make_float3(tex2D(normalMap, texcoord.x, texcoord.y) );
        N = normalize(2 * N - 1);
        N = N.x * tangent_direction 
            + N.y * bitangent_direction 
            + N.z * world_shading_normal;
    }
    N = normalize(N );
    
    float3 hitPoint = ray.origin + t_hit * ray.direction;
    prd_radiance.origin = hitPoint;

    float eta = intIOR / extIOR;
    float cosTheta_i = dot(N, V);
    if(cosTheta_i < 0)
    {
        eta = 1.0 / eta;
        cosTheta_i = - cosTheta_i;
        N = -N;
    }

    float3 refracDirec;
    const bool isTotalReflect = !optix::refract(refracDirec, -V, N, eta );

    prd_radiance.direction = refracDirec;
    
    float3 Z = normalize(-cameraW);
    float3 X = normalize(cameraU);
    float3 Y = normalize(cameraV);
    float3 camN = make_float3( dot(N, X), dot(N, Y), dot(N, Z) );
    float3 camHitPoint = make_float3(
            dot(X, hitPoint - eye ), 
            dot(Y, hitPoint - eye ), 
            dot(Z, hitPoint - eye )
            );
    if(prd_radiance.depth == 0){
        prd_radiance.normal1 = camN;
        prd_radiance.depth1 = camHitPoint;
        prd_radiance.isHit = true;
        prd_radiance.mask1 = 1.0;
    }
    else if(prd_radiance.depth == 1){
        prd_radiance.normal2 = camN;
        prd_radiance.depth2 = camHitPoint;
        if(!isTotalReflect ){
            prd_radiance.mask2 = 1.0;
        }
    }
    
}

// any_hit_shadow program for every material include the lighting should be the same
RT_PROGRAM void any_hit_shadow()
{
    prd_shadow.inShadow = true;
    rtTerminateRay();
}

